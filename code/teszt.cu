    
#include <hip/hip_runtime.h>
#include <iostream>
    #include <math.h>
     
    // CUDA kernel to add elements of two arrays
    __global__
    void add(int n, float *x, float *y)
    {
      int index = blockIdx.x * blockDim.x + threadIdx.x;
      int stride = blockDim.x * gridDim.x;
      for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
    }
     
    int main(void)
    {
      int N = 1<<25;
      float *x, *y;
     
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      // Allocate Unified Memory -- accessible from CPU or GPU
      hipMallocManaged(&x, N*sizeof(float));
      hipMallocManaged(&y, N*sizeof(float));
     
      // initialize x and y arrays on the host
      for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
      }
     
      // Launch kernel on 1M elements on the GPU
      int blockSize = 32;
      int numBlocks = (N + blockSize - 1) / blockSize;
      
      hipEventRecord(start);
      add<<<numBlocks, blockSize>>>(N, x, y);
      hipEventRecord(stop);
     
      // Wait for GPU to finish before accessing on host
      hipDeviceSynchronize();
      hipEventSynchronize(stop);

      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);

      // Check for errors (all values should be 3.0f)
      float maxError = 0.0f;
      for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
      std::cout << "Max error: " << maxError << std::endl;
      std::cout << "Eltelt idő: " << milliseconds << " ms" << std::endl;
     
      // Free memory
      hipFree(x);
      hipFree(y);
     
      return 0;
    }
